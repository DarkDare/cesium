#include "hip/hip_runtime.h"
#include "fastrbf.cu.h"

#undef Success
#include <Eigen/Dense>

#include <common/scoped_ptr.h>
#include <common/types.h>
#include <hip/hip_runtime.h>
#include <glog/logging.h>
#include <stdio.h>
#include <stdlib.h>
#include "util/assert.h"

using Eigen::VectorXf;
using Eigen::MatrixXf;

namespace slib {
  namespace interpolation {
    
    __global__ 
    void Interpolate(float* interpolated, const float* points, const float* w, 
		     const int N, const float alpha, const float epsilon2,
		     const float* samples, const int num_samples) {
      const int index = threadIdx.x + blockIdx.x * blockDim.x;
      if (index < num_samples) {
	const float x = samples[2 * index + 0];
	const float y = samples[2 * index + 1];
	
	float sum = 0.0f;
	for (int32 i = 0; i < N; ++i) {
	  const float d0 = points[2 * i + 0] - x;
	  const float d1 = points[2 * i + 1] - y;
	  const float d = d0 * d0 + d1 * d1;
	  sum += w[i] * sqrt(d + epsilon2);
	}
	
	interpolated[index] = sum + alpha;
      }
    }
    
    Eigen::VectorXf CUDAInterpolatePoints(const MatrixXf& _points, const VectorXf& _w, 
					  const float& _alpha, const float& _epsilon2, 
					  const MatrixXf& _samples) {
      const int D = _samples.cols();
      const int N = _w.size();
      const int num_samples = _samples.rows();
      
      ASSERT_EQ(D, 2);
      
      scoped_array<float> points(new float[N * D]);
      scoped_array<float> w(new float[N]);
      for (int i = 0; i < N; ++i) {
	points[2 * i + 0] = _points(i, 0);
	points[2 * i + 1] = _points(i, 1);
	w[i] = _w(i);
      }
      
      scoped_array<float> samples(new float[num_samples * D]);
      for (int i = 0; i < num_samples; ++i) {
	samples[2 * i + 0] = _samples(i, 0);
	samples[2 * i + 1] = _samples(i, 1);
      }
      
      int threadsPerBlock = 256;
      int blocksPerGrid = (num_samples + threadsPerBlock - 1) / threadsPerBlock;
      VLOG(1) << "CUDA kernel launch with " << blocksPerGrid << " blocks of " << threadsPerBlock << " threads";
      
      float* d_interpolated; 
      CUDA_MALLOC(d_interpolated, sizeof(float) * num_samples);
      
      float* d_points; 
      CUDA_MALLOC(d_points, sizeof(float) * N * D);
      
      float* d_w; 
      CUDA_MALLOC(d_w, sizeof(float) * N);
      
      float* d_samples;
      CUDA_MALLOC(d_samples, sizeof(float) * num_samples * D);
      
      CUDA_UPLOAD(d_points, points.get(), sizeof(float) * N * D);
      CUDA_UPLOAD(d_w, w.get(), sizeof(float) * N);
      CUDA_UPLOAD(d_samples, samples.get(), sizeof(float) * num_samples * 2);
      
      Interpolate<<<blocksPerGrid, threadsPerBlock>>>
	(d_interpolated, d_points, d_w, N, _alpha, _epsilon2, d_samples, num_samples);
      
      // Save a little memory by copying the result into a buffer that is
      // already allocated and large enough to hold the result.
      CUDA_DOWNLOAD(samples.get(), d_interpolated, sizeof(float) * num_samples);
      
      VectorXf interpolated(num_samples);
      for (int i = 0; i < num_samples; ++i) {
	interpolated(i) = samples[i];
      }
      
      return interpolated;
    }
    
  }  // namepsace interpolation
}  // namespace slib
